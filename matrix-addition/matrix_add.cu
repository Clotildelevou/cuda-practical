#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

    __global__
void add_matrix(int *matrixA, int *matrixB, int *matrixC, int size)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index < size)
    {
        printf("Operating index : %d\n", index);
        matrixC[index] = matrixA[index] + matrixB[index];
    }
}

int main(int argc, char *argv[])
{

    if (argc != 2)
        return 1;
    size_t side = atoi(argv[1]);

    size_t size = side * side * sizeof(int);

    //CPU copy of matrix
    int *cpu_a = (int*) malloc(size);
    int *cpu_b = (int*) malloc(size);
    int *cpu_c = (int*) malloc(size);

    //GPU copy of matrix
    int *gpu_a;
    int *gpu_b;
    int *gpu_c;

    hipMalloc((void**) &gpu_a, size);
    hipMalloc((void**) &gpu_b, size);
    hipMalloc((void**) &gpu_c, size);

    //Filling the matrixes
    for (int i = 0; i < pow(side, 2); i++)
    {
        cpu_a[i] = rand() % 100;
        cpu_b[i] = rand() % 100;
    }

    //Copy cpu to gpu device
    hipMemcpy(gpu_a, cpu_a, size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, cpu_b, size, hipMemcpyHostToDevice);

    //Launch matrix_add()
    int thr_per_blk = 256;
    int blk_in_grid = ceil(float(size) / thr_per_blk);
    if (blk_in_grid == 0)
        blk_in_grid = 1;

    add_matrix<<<blk_in_grid, thr_per_blk>>>(gpu_a, gpu_b, gpu_c, side * side);
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
    hipMemcpy(cpu_c, gpu_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < side; i++)
    {
        if (cpu_c[i] != cpu_a[i] + cpu_b[i])
        {
            fprintf(stderr, "Something went wrong...\n mat[%d] is %d and should be %d\n",
                    i, cpu_c[i], cpu_a[i] + cpu_b[i]);
            return 1;
        }
    }
    
    free(cpu_a);
    free(cpu_b);
    free(cpu_c);
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);
    printf("This program is a success !\n");
    return 0;

}
