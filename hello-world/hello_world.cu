#include <cstdio>
#include "hip/hip_runtime.h"

__global__
void GPUFunction()
{
  printf("hello from the Gpu.\n");
}

int main()
{
  GPUFunction<<<1, 1>>>();

  hipDeviceSynchronize();

  return EXIT_SUCCESS;
}
